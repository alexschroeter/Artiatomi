#include "hip/hip_runtime.h"
//Includes for IntelliSense 
#define _SIZE_T_DEFINED
#ifndef __HIPCC__
#define __HIPCC__
#endif
#ifndef __cplusplus
#define __cplusplus
#endif


#define EPS (0.000001f)

#include <hip/hip_runtime.h>
#include <>
#include <>
#include "float.h"
#include <builtin_types.h>
#include <vector_functions.h>

texture<float, 3, hipReadModeElementType> texVol;
texture<float, 3, hipReadModeElementType> texShift;
texture<float2, 3, hipReadModeElementType> texVolCplx;

extern "C"
__global__ void rot3d(int size, float3 rotMat0, float3 rotMat1, float3 rotMat2, float* outVol)
{
	const unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
	const unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;	
	const unsigned int z = blockIdx.z * blockDim.z + threadIdx.z;	

	float center = size / 2;

	float3 vox = make_float3(x - center, y - center, z - center);
	float3 rotVox;
	rotVox.x = center + rotMat0.x * vox.x + rotMat1.x * vox.y + rotMat2.x * vox.z;
	rotVox.y = center + rotMat0.y * vox.x + rotMat1.y * vox.y + rotMat2.y * vox.z;
	rotVox.z = center + rotMat0.z * vox.x + rotMat1.z * vox.y + rotMat2.z * vox.z;

	outVol[z * size * size + y * size + x] = tex3D(texVol, rotVox.x + 0.5f, rotVox.y + 0.5f, rotVox.z + 0.5f);

}

extern "C"
__global__ void rot3dCplx(int size, float3 rotMat0, float3 rotMat1, float3 rotMat2, float2* outVol)
{
	const unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
	const unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;	
	const unsigned int z = blockIdx.z * blockDim.z + threadIdx.z;	
	
	float center = size / 2;

	float3 vox = make_float3(x - center, y - center, z - center);
	float3 rotVox;
	rotVox.x = center + rotMat0.x * vox.x + rotMat1.x * vox.y + rotMat2.x * vox.z;
	rotVox.y = center + rotMat0.y * vox.x + rotMat1.y * vox.y + rotMat2.y * vox.z;
	rotVox.z = center + rotMat0.z * vox.x + rotMat1.z * vox.y + rotMat2.z * vox.z;

	outVol[z * size * size + y * size + x] = tex3D(texVolCplx, rotVox.x + 0.5f, rotVox.y + 0.5f, rotVox.z + 0.5f);
}


extern "C"
	__global__ void shift(int size, float* outVol, float3 shift)
{
	const int x = blockIdx.x * blockDim.x + threadIdx.x;
	const int y = blockIdx.y * blockDim.y + threadIdx.y;	
	const int z = blockIdx.z * blockDim.z + threadIdx.z;	
	
	float sx = float(x - shift.x + 0.5f) / float(size);
	float sy = float(y - shift.y + 0.5f) / float(size);
	float sz = float(z - shift.z + 0.5f) / float(size); 
	
	outVol[z * size * size + y * size + x] = tex3D(texShift, sx, sy, sz);
}


extern "C"
	__global__ void sub(int size, float* inVol, float* outVol, float val)
{
	const unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
	const unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;	
	const unsigned int z = blockIdx.z * blockDim.z + threadIdx.z;	
	
	outVol[z * size * size + y * size + x] = inVol[z * size * size + y * size + x] - val;
}


extern "C"
	__global__ void add(int size, float* inVol, float* outVol)
{
	const unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
	const unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;	
	const unsigned int z = blockIdx.z * blockDim.z + threadIdx.z;	
	
	outVol[z * size * size + y * size + x] += inVol[z * size * size + y * size + x];
}


extern "C"
__global__ void subCplx(int size, float2* inVol, float2* outVol, float* subval, float divVal)
{
	const unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
	const unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;	
	const unsigned int z = blockIdx.z * blockDim.z + threadIdx.z;	
	
	float2 temp = inVol[z * size * size + y * size + x];
	temp.x -= subval[0] / divVal;
	outVol[z * size * size + y * size + x] = temp;
}


extern "C"
__global__ void wedgeNorm(int size, float* wedge, float2* part, float* maxVal, int newMethod)
{
	const unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
	const unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;	
	const unsigned int z = blockIdx.z * blockDim.z + threadIdx.z;	
	
	float val = wedge[z * size * size + y * size + x];

	if (newMethod)
	{
		if (val <= 0)
			val = 0;
		else
			val = 1.0f / val;
	}
	else
	{
		if (val < 0.1f * maxVal[0])
			val = 1.0f / (0.1f * maxVal[0]);
		else
			val = 1.0f / val;
	}
	float2 p = part[z * size * size + y * size + x];
	p.x *= val;
	p.y *= val;
	part[z * size * size + y * size + x] = p;
}


extern "C"
__global__ void subCplx2(int size, float2* inVol, float2* outVol, float* subval, float* divVal)
{
	const unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
	const unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;	
	const unsigned int z = blockIdx.z * blockDim.z + threadIdx.z;	
	
	float2 temp = inVol[z * size * size + y * size + x];
	temp.x -= subval[0] / divVal[0];
	outVol[z * size * size + y * size + x] = temp;
}


extern "C"
__global__ void makeReal(int size, float2* inVol, float* outVol)
{
	const unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
	const unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;	
	const unsigned int z = blockIdx.z * blockDim.z + threadIdx.z;	
	
	float2 temp = inVol[z * size * size + y * size + x];
	outVol[z * size * size + y * size + x] = temp.x;
}


extern "C"
__global__ void makeCplxWithSub(int size, float* inVol, float2* outVol, float val)
{
	const unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
	const unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;	
	const unsigned int z = blockIdx.z * blockDim.z + threadIdx.z;	
	
	float2 temp = make_float2(inVol[z * size * size + y * size + x] - val, 0);
	outVol[z * size * size + y * size + x] = temp;
}


extern "C"
__global__ void makeCplxWithSquareAndSub(int size, float* inVol, float2* outVol, float val)
{
	const unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
	const unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;	
	const unsigned int z = blockIdx.z * blockDim.z + threadIdx.z;	
	
	float2 temp = make_float2((inVol[z * size * size + y * size + x] - val) * (inVol[z * size * size + y * size + x] - val), 0);
	outVol[z * size * size + y * size + x] = temp;
}


extern "C"
__global__ void binarize(int size, float* inVol, float* outVol)
{
	const unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
	const unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;	
	const unsigned int z = blockIdx.z * blockDim.z + threadIdx.z;	
	
	outVol[z * size * size + y * size + x] = inVol[z * size * size + y * size + x] > 0.5f ? 1.0f : 0.0f;
}



extern "C"
__global__ void mulVol(int size, float* inVol, float2* outVol)
{
	const unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
	const unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;	
	const unsigned int z = blockIdx.z * blockDim.z + threadIdx.z;	
	
	float2 temp = outVol[z * size * size + y * size + x];
	temp.x *= inVol[z * size * size + y * size + x];
	temp.y *= inVol[z * size * size + y * size + x];
	outVol[z * size * size + y * size + x] = temp;
}



extern "C"
__global__ void mulVolCplx(int size, float2* inVol, float2* outVol)
{
	const unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
	const unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;	
	const unsigned int z = blockIdx.z * blockDim.z + threadIdx.z;	
	
	float2 temp = outVol[z * size * size + y * size + x];
	float2 temp2 = inVol[z * size * size + y * size + x];
	temp.x *= temp2.x; //complex component is meant to be zero
	temp.y *= temp2.x;
	outVol[z * size * size + y * size + x] = temp;
}



extern "C"
__global__ void mul(int size, float in, float2* outVol)
{
	const unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
	const unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;	
	const unsigned int z = blockIdx.z * blockDim.z + threadIdx.z;	
	
	float2 temp = outVol[z * size * size + y * size + x];
	temp.x *= in;
	temp.y *= in;
	outVol[z * size * size + y * size + x] = temp;
}



extern "C"
__global__ void conv(int size, float2* inVol, float2* outVol)
{
	const unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
	const unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;	
	const unsigned int z = blockIdx.z * blockDim.z + threadIdx.z;	
	
	float2 o = outVol[z * size * size + y * size + x];
	float2 i = inVol[z * size * size + y * size + x];
	float2 erg;
	erg.x = (o.x * i.x) - (o.y * i.y);
	erg.y = (o.x * i.y) + (o.y * i.x);
	outVol[z * size * size + y * size + x] = erg;
}



extern "C"
__global__ void correl(int size, float2* inVol, float2* outVol)
{
	const unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
	const unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;	
	const unsigned int z = blockIdx.z * blockDim.z + threadIdx.z;	
	
	float2 o = outVol[z * size * size + y * size + x];
	float2 i = inVol[z * size * size + y * size + x];
	float2 erg;
	erg.x = (o.x * i.x) + (o.y * i.y);
	erg.y = (o.x * i.y) - (o.y * i.x);
	outVol[z * size * size + y * size + x] = erg;
}


extern "C"
__global__ void bandpass(int size, float2* vol, float rDown, float rUp, float smooth)
{
	const unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
	const unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;	
	const unsigned int z = blockIdx.z * blockDim.z + threadIdx.z;	
	
	float2 temp = vol[z * size * size + y * size + x];

	//use squared smooth for Gaussian
	smooth = smooth * smooth;

	float center = size / 2;
	float3 vox = make_float3(x - center, y - center, z - center);

	float dist = sqrt(vox.x * vox.x + vox.y * vox.y + vox.z * vox.z);
	float scf = (dist - rUp) * (dist - rUp);
	smooth > 0 ? scf = exp(-scf/smooth) : scf = 0;

	if (dist > rUp)
	{
		temp.x *= scf;
		temp.y *= scf;
	}
	
	scf = (dist - rDown) * (dist - rDown);
	smooth > 0 ? scf = exp(-scf/smooth) : scf = 0;
	
	if (dist < rDown)
	{
		temp.x *= scf;
		temp.y *= scf;
	}
	

	vol[z * size * size + y * size + x] = temp;
}


extern "C"
__global__ void bandpassFFTShift(int size, float2* vol, float rDown, float rUp, float smooth)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;	
	int z = blockIdx.z * blockDim.z + threadIdx.z;	
	
	int i = (x + size / 2) % size;
	int j = (y + size / 2) % size;
	int k = (z + size / 2) % size;

	float2 temp = vol[z * size * size + y * size + x];

	//use squared smooth for Gaussian
	smooth = smooth * smooth;

	float center = size / 2;
	float3 vox = make_float3(i - center, j - center, k - center);

	float dist = sqrt(vox.x * vox.x + vox.y * vox.y + vox.z * vox.z);
	float scf = (dist - rUp) * (dist - rUp);
	smooth > 0 ? scf = exp(-scf/smooth) : scf = 0;

	if (dist > rUp)
	{
		temp.x *= scf;
		temp.y *= scf;
	}
	
	scf = (dist - rDown) * (dist - rDown);
	smooth > 0 ? scf = exp(-scf/smooth) : scf = 0;
	
	if (dist < rDown)
	{
		temp.x *= scf;
		temp.y *= scf;
	}
	

	vol[z * size * size + y * size + x] = temp;
}


extern "C"
__global__ void fftshift(int size, float2* vol)
{
	const unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
	const unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;	
	const unsigned int z = blockIdx.z * blockDim.z + threadIdx.z;	
	
	float2 temp = vol[z * size * size + y * size + x]; 
	
	int mx = x - size / 2;
	int my = y - size / 2;
	int mz = z - size / 2;

	float a = 1.0f - 2 * (((mx + my + mz) & 1));
	
	temp.x *= a;
	temp.y *= a;

	vol[z * size * size + y * size + x] = temp;
}


extern "C"
__global__ void fftshift2(int size, float2* volIn, float2* volOut)
{
	const int x = blockIdx.x * blockDim.x + threadIdx.x;
	const int y = blockIdx.y * blockDim.y + threadIdx.y;	
	const int z = blockIdx.z * blockDim.z + threadIdx.z;	
	
	int i = (x + size / 2) % size;
	int j = (y + size / 2) % size;
	int k = (z + size / 2) % size;


	float2 temp = volIn[k * size * size + j * size + i]; 
	volOut[z * size * size + y * size + x] = temp;
}



extern "C"
__global__ void fftshiftReal(int size, float* volIn, float* volOut)
{
	const int x = blockIdx.x * blockDim.x + threadIdx.x;
	const int y = blockIdx.y * blockDim.y + threadIdx.y;	
	const int z = blockIdx.z * blockDim.z + threadIdx.z;	
	
	int i = (x + size / 2) % size;
	int j = (y + size / 2) % size;
	int k = (z + size / 2) % size;


	float temp = volIn[k * size * size + j * size + i]; 
	volOut[z * size * size + y * size + x] = temp;
}



extern "C"
__global__ void energynorm(int size, float2* particle, float2* partSqr, float2* cccMap, float* energyRef, float* nVox)
{
	const unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
	const unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;	
	const unsigned int z = blockIdx.z * blockDim.z + threadIdx.z;	
	
	float part = particle[z * size * size + y * size + x].x; 
	float energyLocal = partSqr[z * size * size + y * size + x].x; 
	
	float2 erg;
	erg.x = 0;
	erg.y = 0;

	energyLocal -= part * part / nVox[0];
	energyLocal = sqrt(energyLocal) * sqrt(energyRef[0]);

	if (energyLocal > EPS)
	{
		erg.x = cccMap[z * size * size + y * size + x].x / energyLocal;
	}

	cccMap[z * size * size + y * size + x] = erg;
}





extern "C"
__global__ void findmax(float* maxVals, float* index, float* val, float rphi, float rpsi, float rthe)
{
	float oldmax = maxVals[0];
	if (val[0] > oldmax)
	{
		maxVals[0] = val[0];
		maxVals[1] = index[0];
		maxVals[2] = rphi;
		maxVals[3] = rpsi;
		maxVals[4] = rthe;
	}
}


