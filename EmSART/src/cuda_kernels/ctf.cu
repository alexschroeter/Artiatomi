#include "hip/hip_runtime.h"
//  Copyright (c) 2018, Michael Kunz and Frangakis Lab, BMLS,
//  Goethe University, Frankfurt am Main.
//  All rights reserved.
//  http://kunzmi.github.io/Artiatomi
//  
//  This file is part of the Artiatomi package.
//  
//  Artiatomi is free software: you can redistribute it and/or modify
//  it under the terms of the GNU General Public License as published by
//  the Free Software Foundation, either version 3 of the License, or
//  (at your option) any later version.
//  
//  Artiatomi is distributed in the hope that it will be useful,
//  but WITHOUT ANY WARRANTY; without even the implied warranty of
//  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
//  GNU General Public License for more details.
//  
//  You should have received a copy of the GNU General Public License
//  along with Artiatomi. If not, see <http://www.gnu.org/licenses/>.
//  
////////////////////////////////////////////////////////////////////////


#ifndef CTF_CU
#define CTF_CU


//Includes for IntelliSense 
#define _SIZE_T_DEFINED
#ifndef __HIPCC__
#define __HIPCC__
#endif
#ifndef __cplusplus
#define __cplusplus
#endif
#include <hip/hip_runtime.h>
#include <>
#include <>
#include "float.h"
#include "math.h"
#include "hipfft/hipfft.h"
#include <builtin_types.h>
#include <vector_functions.h>

#define M_PI       3.14159265358979323846f
//#define _voltage (300.0f)
#define h ((float)6.63E-34) //Planck's quantum
#define c ((float)3.00E+08) //Light speed
#define Cs (c_cs * 0.001f)
#define Cc (c_cs * 0.001f)
				
#define PhaseShift (0)
#define EnergySpread (0.7f) //eV
#define E0 (511) //keV
#define RelativisticCorrectionFactor ((1 + c_voltage / (E0 * 1000))/(1 + ((c_voltage*1000) / (2 * E0 * 1000))))
#define H ((Cc * EnergySpread * RelativisticCorrectionFactor) / (c_voltage * 1000))

#define a1 (1.494f) //Scat.Profile Carbon Amplitude 1
#define a2 (0.937f) //Scat.Profile Carbon Amplitude 2
#define b1 (23.22f * (float)1E-20) //Scat.Profile Carbon Halfwidth 1
#define b2 (3.79f * (float)1E-20)  //Scat.Profile Carbon Halfwidth 2

#define lambda ((h * c) / sqrtf(((2 * E0 * c_voltage * 1000.0f * 1000.0f) + (c_voltage * c_voltage * 1000.0f * 1000.0f)) * 1.602E-19 * 1.602E-19))

__device__ __constant__ float c_cs;
__device__ __constant__ float c_voltage;
__device__ __constant__ float c_openingAngle;
__device__ __constant__ float c_ampContrast;
__device__ __constant__ float c_phaseContrast;
__device__ __constant__ float c_pixelsize;
__device__ __constant__ float c_pixelcount;
__device__ __constant__ float c_maxFreq;
__device__ __constant__ float c_freqStepSize;
//__device__ __constant__ float c_lambda;
__device__ __constant__ float c_applyScatteringProfile;
__device__ __constant__ float c_applyEnvelopeFunction;

// transform vector by matrix

extern "C"
__global__ 
void ctf(hipComplex* ctf, size_t stride, float defocusMin, float defocusMax, float angle, bool applyForFP, bool phaseFlipOnly, float WienerFilterNoiseLevel, float4 betaFac)
{
	//compute x,y,z indiced
	unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;	
	unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
	
	if (x >= c_pixelcount/2 + 1) return;
	if (y >= c_pixelcount) return;

			
	//float length = sqrtf((x-c_pixelcount/2) * (x-c_pixelcount/2) + (y-c_pixelcount/2) * (y-c_pixelcount/2));
	float xpos = x;
	float ypos = y;//-c_pixelcount/2;
	if (ypos > c_pixelcount * 0.5f)
		ypos = (c_pixelcount - ypos) * -1.0f;
	
	float alpha;
	if (xpos == 0)
		alpha = (M_PI * 0.5f);
	else
		alpha = (atan2(ypos , xpos));
		
	float beta = ((alpha - angle));
	
	float def0 = defocusMin;
	float def1 = defocusMax;

	float defocus = def0 + (1 - cos(2*beta)) * (def1 - def0);

	float length = sqrtf(xpos * xpos + ypos * ypos);

	//float angle = 76.0f / M_PI * 180.0f;
	//float stretchX = 1.066f;
	/*float angle = 80.0f / M_PI * 180.0f;
	float stretchX = 1.026f;

	float temp = cos(angle) * xpos - sin(angle) * ypos;
	ypos = sin(angle) * xpos + cos(angle) * ypos;
	xpos = temp;
	xpos *= stretchX;*/
	
	//float w = 1.0f;
	//if (length > c_pixelcount * 0.5f)
	//{
	//	w = (length - c_pixelcount * 0.1f) / (c_pixelcount - c_pixelcount * 0.1f) ;
	//	w = expf(-(w * w * 9 * 9));
	//}
    length *= c_freqStepSize;

    float o = expf(-14.238829f * (c_openingAngle * c_openingAngle * ((Cs * lambda * lambda * length * length * length - defocus * length) * (Cs * lambda * lambda * length * length * length - defocus * length))));
    float p = expf(-((0.943359f * lambda * length * length * H) * (0.943359f * lambda * length * length * H)));
    float q = (a1 * expf(-b1 * (length * length)) + a2 * expf(-b2 * (length * length))) / 2.431f;

    float m = -PhaseShift + (M_PI / 2.0f) * (Cs * lambda * lambda * lambda * length * length * length * length - 2 * defocus * lambda * length * length);
    float n = c_phaseContrast * sinf(m) + c_ampContrast * cosf(m);

    //float r = (o * p * ((1 - c_applyScatteringProfile) + (q * c_applyScatteringProfile))) * c_applyEnvelopeFunction + (1 - c_applyEnvelopeFunction);

    hipComplex res = ctf[y * stride / sizeof(hipComplex) + x];
//res.x = 1;res.y=0;
    //float limit1 = 4084-(sinf(5.5f * 3.14159265f / 180.0f) * x);
    //float limit2 = 4095-(sinf(1.5f * 3.14159265f / 180.0f) * x);
	
    if (applyForFP && sqrtf(xpos * xpos + ypos * ypos) > betaFac.x && !phaseFlipOnly)// && length < 317382812)
    {
		//double faq = coeefs[0] * Math.Exp(-(Math.Abs(coeefs[1])) * freq - coeefs[2] * freq * freq - coeefs[3] * freq * freq * freq);
		//const float coeff0 = 1.0f;
		length = length / 100000000.0f;
		/*const float coeff1 = 0.0000000000000001f;
		const float coeff2 = -0.00652468f;
		const float coeff3 = 0.003948657f;*/
		/*const float coeff1 = 0;
		const float coeff2 = 0.008f;
		const float coeff3 = 0;*/
		float coeff1 = betaFac.y;
		float coeff2 = betaFac.z;
		float coeff3 = betaFac.w;
		float expfun = expf((-coeff1 * length - coeff2 * length * length - coeff3 * length * length * length));
		expfun = max(expfun, 0.01f);
		float val = n * expfun;
		//val = fmaxf(val, 0.005f);
		if (abs(val) < 0.0001f && val >=0 ) val = 0.0001f;
		if (abs(val) < 0.0001f && val < 0 ) val = -0.0001f;
		
		
		res.x = res.x * -val;
		res.y = res.y * -val;
		//res.x *= val;
		//res.y *= val;
    }
  //   if (n > 0)//)// && y <= limit2 && length < 317382812)&& y < 4040 && (y <= limit1 && y > 1)
  //  {
		//res.x *= -1.0f;
		//res.y *= -1.0f;
  //  }
    /*if (n > 0 && !absolut)//)// && y <= limit2 && length < 317382812)&& y < 4040 && (y <= limit1 && y > 1)
    {
		res.x *= -1.0f;
		res.y *= -1.0f;
    }*/
	//else
	//{
	//	/*res.x = 0;
	//	res.y = 0;*/
	//}
	//res.x = 1;
    if (!applyForFP && sqrtf(xpos * xpos + ypos * ypos) > betaFac.x && !phaseFlipOnly)// && length < 317382812)
    {
		//double faq = coeefs[0] * Math.Exp(-(Math.Abs(coeefs[1])) * freq - coeefs[2] * freq * freq - coeefs[3] * freq * freq * freq);
		//const float coeff0 = 1.0f;
		length = length / 100000000.0f;
		float coeff1 = betaFac.y;
		float coeff2 = betaFac.z;
		float coeff3 = betaFac.w;
		float expfun = expf((-coeff1 * length - coeff2 * length * length - coeff3 * length * length * length));
		expfun = max(expfun, WienerFilterNoiseLevel);
		float val = n * expfun;
		//val = fmaxf(val, 0.005f);
		/*if (abs(val) < 0.00001f && val >=0 ) val = 0.00001f;
		if (abs(val) < 0.00001f && val < 0 ) val = -0.00001f;*/
		
		
		res.x = res.x * -val / (val * val + WienerFilterNoiseLevel);
		res.y = res.y * -val / (val * val + WienerFilterNoiseLevel);
		//res.x = expfun;
		//res.y = expfun;
    }
    
	if (phaseFlipOnly)
	{
		if (n >= 0)
		{
			res.x = -res.x;
			res.y = -res.y;
		}
	}
    
    //else
    //{
	//if (abs(r*n) > 0.5f)
	//{
	    //res.x /= abs(r*n);
	    //res.y /= abs(r*n);
        //}
    //}
    //res.x = 1;
    //if (absolut)
    //    res.y = r * n * w;
    //else
    //{
        //res.y = r * n;
        //if (r * n < 0) //res.y = 1.0f;
        //else 
	//res.y *= -1.0f;	
    //res.x = 0; res.y = 0;
    ctf[y * stride / sizeof(hipComplex) + x] = res;
    //}
//res.x = w;
}

#endif
